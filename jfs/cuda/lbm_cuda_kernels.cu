#include "hip/hip_runtime.h"
#include "lbm_cuda_kernels.h"

#include <jfs/cuda/grid/cuda_grid2d.h>

namespace jfs {

    using FieldType2D::Vector;
    using FieldType2D::Scalar;

    __constant__ float cs = 0.57735026919;
    __constant__ float lat_uref = .2;
    __constant__ int c[9][2] { // D2Q9 velocity discretization
        {0,0},                                // i = 0
        {1,0}, {-1,0}, {0,1}, {0,-1},   // i = 1, 2, 3, 4
        {1,1}, {-1,1}, {1,-1}, {-1,-1}  // i = 5, 6, 7, 8
    };

    __constant__ int bounce_back_indices[9]{
        0,
        2, 1, 4, 3,
        8, 7, 6, 5
    };

    __constant__ float w[9] = { // lattice weights
        4./9.,                          // i = 0
        1./9., 1./9., 1./9., 1./9.,     // i = 1, 2, 3, 4
        1./36., 1./36., 1./36., 1./36., // i = 5, 6, 7, 8
    };

    __constant__ LBMSolverProps const_props[1]{};
    CudaLBMSolver* current_cuda_lbm_solver = nullptr;

/*
*
DEVICE FUNCTIONS
*
*/
// alpha represents the lattice index
    __device__
    float calcEquilibrium(int alpha, int i, int j) {
#ifdef __CUDA_ARCH__
        float ci[2]{(float) c[alpha][0], (float) c[alpha][1]};

        const float &rho = *(const_props[0].rho_grid + const_props[0].grid_size * 1 * j + 1 * i);

        const float *u = const_props[0].u_grid + const_props[0].grid_size * 2 * j + 2 * i;

        float &rho0 = const_props[0].rho0;
        float &uref = const_props[0].uref;

        float u_cpy[2]{u[0], u[1]};
        u_cpy[0] *= lat_uref / uref;
        u_cpy[1] *= lat_uref / uref;

        float ci_dot_u = ci[0] * u_cpy[0] + ci[1] * u_cpy[1];
        float u_dot_u = u_cpy[0] * u_cpy[0] + u_cpy[1] * u_cpy[1];

        return w[alpha] * rho / rho0 *
               (1 + ci_dot_u / (powf(1 / sqrtf(3), 2)) + powf(ci_dot_u, 2) / (2 * powf(1 / sqrtf(3), 4)) -
                u_dot_u / (2 * powf(1 / sqrtf(3), 2)));
#endif
    }

    __device__
    float calcLatticeForce(int alpha, int i, int j) {
#ifdef __CUDA_ARCH__
        const float ci[2]{(float) c[alpha][0], (float) c[alpha][1]};

        const float &rho = *(const_props[0].rho_grid + const_props[0].grid_size * 1 * j + 1 * i);

        const float *u = const_props[0].u_grid + const_props[0].grid_size * 2 * j + 2 * i;

        const float *force = const_props[0].force_grid + const_props[0].grid_size * 2 * j + 2 * i;

        const float &rho0 = const_props[0].rho0;
        const float dx = const_props[0].grid_length / ((float) const_props[0].grid_size - 1.f);
        const float &uref = const_props[0].uref;

        float force_cpy[2]{force[0], force[1]};
        force_cpy[0] *= 1 / rho0 * dx * powf(lat_uref / uref, 2);
        force_cpy[1] *= 1 / rho0 * dx * powf(lat_uref / uref, 2);

        float u_cpy[2]{u[0], u[1]};
        u_cpy[0] *= lat_uref / uref;
        u_cpy[1] *= lat_uref / uref;

        float ci_dot_u = ci[0] * u_cpy[0] * +ci[1] * u_cpy[1];

        return (1 - const_props[0].lat_tau / 2) * w[alpha] * (
                ((1 / powf(cs, 2)) * (ci[0] - u_cpy[0]) + (ci_dot_u / powf(cs, 4)) * ci[0]) * force_cpy[0] +
                ((1 / powf(cs, 2)) * (ci[1] - u_cpy[1]) + (ci_dot_u / powf(cs, 4)) * ci[1]) * force_cpy[1]
        );
#endif
    }

    __device__
    void calcPhysicalProps(int i, int j) {
#ifdef __CUDA_ARCH__

        const float *f = const_props[0].f_grid + const_props[0].grid_size * 9 * j + 9 * i;

        float &rho = *(const_props[0].rho_grid + const_props[0].grid_size * 1 * j + 1 * i);

        float *u = const_props[0].u_grid + const_props[0].grid_size * 2 * j + 2 * i;

        float &rho0 = const_props[0].rho0;
        float &uref = const_props[0].uref;

        u[0] = 0;
        u[1] = 0;
        rho = 0;
        for (int alpha = 0; alpha < 9; alpha++) {
            rho += f[alpha];
            u[0] += (float) c[alpha][0] * f[alpha];
            u[1] += (float) c[alpha][1] * f[alpha];
        }
        u[0] = uref / lat_uref * (u[0] / rho);
        u[1] = uref / lat_uref * (u[1] / rho);
        rho *= rho0;
#endif
    }

/*
*
END DEVICE FUNCTIONS
*
*/
    __global__
    void forceVelocityKernel(int i, int j, float ux, float uy) {

#ifdef __CUDA_ARCH__
        CudaGrid2D<Vector> u_grid;
        u_grid.MapData(const_props[0].u_grid, const_props[0].grid_size, 1);
        CudaGrid2D<Vector> force_grid;
        force_grid.MapData(const_props[0].force_grid, const_props[0].grid_size, 1);
        CudaGrid2D<Scalar> rho_grid;
        rho_grid.MapData(const_props[0].rho_grid, const_props[0].grid_size, 1);

        force_grid(i, j, 0, 0) += (ux - u_grid(i, j, 0, 0)) * rho_grid(i, j, 0, 0) / const_props[0].dt;
        force_grid(i, j, 0, 1) += (uy - u_grid(i, j, 0, 1)) * rho_grid(i, j, 0, 0) / const_props[0].dt;
#endif
    }

    __global__
    void resetDistributionKernel(float *f_data) {

        int grid_size = const_props[0].grid_size;
        int alpha = blockIdx.x * blockDim.x + threadIdx.x;
        int j = alpha / (grid_size * 9);
        alpha -= grid_size * 9 * j;
        int i = alpha / (9);
        alpha -= 9 * i;

#ifdef __CUDA_ARCH__

        if (i >= grid_size || j >= grid_size)
            return;

        float *f = f_data + grid_size * 9 * j + 9 * i;

        f[alpha] = calcEquilibrium(alpha, i, j);
#endif
    }

    __global__
    __launch_bounds__(256, 6)

    void collideKernel() {

        int grid_size = const_props[0].grid_size;
        int alpha = blockIdx.x * blockDim.x + threadIdx.x;
        int j = alpha / (grid_size * 9);
        alpha -= grid_size * 9 * j;
        int i = alpha / (9);
        alpha -= 9 * i;

#ifdef __CUDA_ARCH__

        if (i >= grid_size || j >= grid_size)
            return;

        float *f = const_props[0].f_grid + grid_size * 9 * j + 9 * i;

        float lat_force = calcLatticeForce(alpha, i, j);
        float fbar = calcEquilibrium(alpha, i, j);

        f[alpha] += lat_force - (f[alpha] - fbar) / const_props[0].lat_tau;
        *(const_props[0].f0_grid + grid_size * 9 * j + 9 * i + alpha) = f[alpha];
#endif
    }

    __global__
    void streamKernel() {

        int grid_size = const_props[0].grid_size;
        int alpha = blockIdx.x * blockDim.x + threadIdx.x;
        int j = alpha / (grid_size * 9);
        alpha -= grid_size * 9 * j;
        int i = alpha / (9);
        alpha -= 9 * i;

#ifdef __CUDA_ARCH__

        if (i >= grid_size || j >= grid_size)
            return;

        int cix = c[alpha][0];
        int ciy = c[alpha][1];

        float *f = const_props[0].f_grid + grid_size * 9 * j + 9 * i;

        if ((j - ciy) >= 0 && (j - ciy) < grid_size && (i - cix) >= 0 && (i - cix) < grid_size) {
            float *f0 = const_props[0].f0_grid + grid_size * 9 * (j - ciy) + 9 * (i - cix);
            f[alpha] = f0[alpha];
        } else {
            float *f0 = const_props[0].f0_grid + grid_size * 9 * j + 9 * i;
            int alpha_bounce = bounce_back_indices[alpha];
            f[alpha] = f0[alpha_bounce];
        }
//        if ( isnan(f[alpha]) || isinf(f[alpha]) )
//            const_props[0].failed_step = true;
#endif
    }

    __global__
    void calcPhysicalKernel() {

        int grid_size = const_props[0].grid_size;
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = i / grid_size;
        i -= j * grid_size;

#ifdef __CUDA_ARCH__
        calcPhysicalProps(i, j);
#endif
    }

    __global__
    void boundaryDampKernel() {


        int grid_size = const_props[0].grid_size;
        int j = blockIdx.x * blockDim.x + threadIdx.x;

        if (j >= grid_size)
            return;

#ifdef __CUDA_ARCH__
        for (int i = 0; i < grid_size; i += (grid_size - 1)) {
            int step;
            if (i == 0)
                step = 1;
            else
                step = -1;

            i += step;

            const_props[0].u_grid[grid_size * 2 * j + 2 * (i - step) + 0] = const_props[0].u_grid[grid_size * 2 * j +
                                                                                                  2 * i + 0];
            const_props[0].u_grid[grid_size * 2 * j + 2 * (i - step) + 1] = const_props[0].u_grid[grid_size * 2 * j +
                                                                                                  2 * i + 1];
            const_props[0].rho_grid[grid_size * j + (i - step)] = const_props[0].rho_grid[grid_size * j + i];

            const_props[0].u_grid[grid_size * 2 * (i - step) + 2 * j + 0] = const_props[0].u_grid[grid_size * 2 * i +
                                                                                                  2 * j + 0];
            const_props[0].u_grid[grid_size * 2 * (i - step) + 2 * j + 1] = const_props[0].u_grid[grid_size * 2 * i +
                                                                                                  2 * j + 1];
            const_props[0].rho_grid[grid_size * (i - step) + j] = const_props[0].rho_grid[grid_size * i + j];

            i -= step;

            for (int alpha = 0; alpha < 9; alpha++) {
                const_props[0].f_grid[grid_size * 9 * j + 9 * i + alpha] = calcEquilibrium(alpha, i, j);
                const_props[0].f_grid[grid_size * 9 * i + 9 * j + alpha] = calcEquilibrium(alpha, j, i);
            }
        }
#endif
    }

} // namespace jfs