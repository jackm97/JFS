#include "hip/hip_runtime.h"
#include "lbm_solver_cuda.h"

#include <jfs/cuda/lbm_cuda_kernels.h>

#include <cstring>
#include <iostream>

namespace jfs {

JFS_INLINE CudaLBMSolver::CudaLBMSolver(ushort grid_size, float grid_length, BoundType btype, float rho0, float visc, float uref) :
cs_{ 1/sqrtf(3) }
{
    Initialize(grid_size, grid_length, btype, rho0, visc, uref);
}

JFS_INLINE void CudaLBMSolver::Initialize(ushort grid_size, float grid_length, BoundType btype, float rho0, float visc, float uref)
{
    grid_size_ = grid_size;
    grid_length_ = grid_length;
    btype_ = btype;

    rho0_ = rho0;
    visc_ = visc;
    uref_ = uref;
    
    // lattice scaling stuff
    us_ = cs_/lat_uref_ * uref_;
    
    dx_ = grid_length_ / (float)grid_size_;
    lat_visc_ = lat_uref_/(uref_ * dx_) * visc;
    lat_tau_ = (3.f * lat_visc_ + .5f);
    dt_ = lat_uref_/uref_ * dx_ * lat_dt_;

    f_grid_.Resize(grid_size_, 9);
    f0_grid_.Resize(grid_size_, 9);

    rho_grid_.Resize(grid_size_, 1);
    rho_grid_mapped_.Resize(grid_size_, 3);

    u_grid_.Resize(grid_size_, 1);

    force_grid_.Resize(grid_size_, 1);

    LBMSolverProps props = SolverProps();
    hipMemcpyToSymbol(HIP_SYMBOL(const_props), &props, sizeof(LBMSolverProps), 0, hipMemcpyHostToDevice);
    current_cuda_lbm_solver = this;
    hipDeviceSynchronize();

    ResetFluid();
}

JFS_INLINE void CudaLBMSolver::ResetFluid()
{
    for (int d = 0; d < 2; d++)
    {
        u_grid_.SetGridToValue(0, 0, d);
        force_grid_.SetGridToValue(0, 0, d);
    }
    rho_grid_.SetGridToValue(rho0_, 0, 0);

    int threads_per_block = 256;
    int num_blocks = (9*(int)grid_size_*(int)grid_size_) / threads_per_block + 1;
    resetDistributionKernel <<<num_blocks, threads_per_block>>> (f_grid_.Data());
    hipDeviceSynchronize();

    time_ = 0;
}

JFS_INLINE bool CudaLBMSolver::CalcNextStep(const std::vector<Force>& forces)
{
    bool failedStep = false;
    try
    {   
        for (const auto & i : forces)
        {
            float force[3] = {
                i.force[0],
                i.force[1],
                i.force[2]
            };
            float point[3] = {
                i.pos[0]/grid_length_ * (float)grid_size_,
                i.pos[1]/grid_length_ * (float)grid_size_,
                i.pos[2]/grid_length_ * (float)grid_size_
            };
            if (point[0] < (float)grid_size_ && point[0] >= 0 && point[1] < (float)grid_size_ && point[1] >= 0)
                for (int d = 0; d < 2; d++)
                {
                    force_grid_.InterpToGrid(force[d], point[0], point[1], 0, d);
                }
        }
        
        failedStep = CalcNextStep();
    }
    catch(const std::exception& e)
    {
        std::cerr << e.what() << '\n';
        failedStep = true;
    }
    for (int d = 0; d < 2; d++)
    {
        force_grid_.SetGridToValue(0, 0, d);
    }

    if (failedStep) ResetFluid();

    return failedStep;
}

JFS_INLINE void CudaLBMSolver::ForceVelocity(ushort i, ushort j, float ux, float uy)
{
    if (current_cuda_lbm_solver != this)
    {
        LBMSolverProps props = SolverProps();
        hipMemcpyToSymbol(HIP_SYMBOL(const_props), &props, sizeof(LBMSolverProps), 0, hipMemcpyHostToDevice);
        current_cuda_lbm_solver = this;
    }

    forceVelocityKernel <<<1, 1>>> (i, j, ux, uy);
    hipDeviceSynchronize();
}

JFS_INLINE void CudaLBMSolver::SetDensityMapping(float minrho, float maxrho)
{
    min_rho_ = minrho;
    max_rho_ = maxrho;
}

JFS_INLINE void CudaLBMSolver::DensityExtrema(float minmax_rho[2])
{
    float* rho_grid_host = rho_grid_.HostData();

    float min_rho = rho_grid_host[0];
    float max_rho = rho_grid_host[0];

    for (int i=0; i < grid_size_*grid_size_; i++)
    {
        if (rho_grid_host[i] < min_rho)
            min_rho = rho_grid_host[i];
    }

    for (int i=0; i < grid_size_*grid_size_; i++)
    {
        if (rho_grid_host[i] > max_rho)
            max_rho = rho_grid_host[i];
    }

    minmax_rho[0] = min_rho;
    minmax_rho[1] = max_rho;
}

JFS_INLINE bool CudaLBMSolver::CalcNextStep()
{
    LBMSolverProps props{};
    if (current_cuda_lbm_solver != this)
    {
        props = SolverProps();
        hipMemcpyToSymbol(HIP_SYMBOL(const_props), &props, sizeof(LBMSolverProps), 0, hipMemcpyHostToDevice);
        current_cuda_lbm_solver = this;
    }

    int threads_per_block = 256;
    int num_blocks = (9*(int)grid_size_*(int)grid_size_) / threads_per_block + 1;
    
    resetDistributionKernel <<<num_blocks, threads_per_block>>> (f0_grid_.Data());
    collideKernel <<<num_blocks, threads_per_block>>> ();
    hipDeviceSynchronize();

    streamKernel <<<num_blocks, threads_per_block>>> ();
    hipDeviceSynchronize();

    calcPhysicalKernel <<<num_blocks/9, threads_per_block>>> ();
    hipDeviceSynchronize();

    // do any field manipulations before next step
    if (btype_ == DAMPED)
    {
        boundaryDampKernel <<<grid_size_/threads_per_block + 1, threads_per_block>>> ();
    }

    time_ += dt_;

    hipMemcpyFromSymbol(&props, HIP_SYMBOL(const_props), sizeof(LBMSolverProps), 0, hipMemcpyDeviceToHost);
    return props.failed_step;
}

__host__
JFS_INLINE void CudaLBMSolver::MapDensity()
{
    float* host_rho_grid = RhoData();

    float min_rho = host_rho_grid[0];
    float max_rho = host_rho_grid[0];
    float mean_rho = 0;
    for (int i = 0; i < grid_size_*grid_size_; i++)
        mean_rho += host_rho_grid[i];
    mean_rho /= (float)grid_size_ * (float)grid_size_;

    for (int i=0; i < grid_size_*grid_size_ && min_rho_ == -1; i++)
    {
        if (host_rho_grid[i] < min_rho)
            min_rho = host_rho_grid[i];
    }

    if (min_rho_ != -1)
        min_rho = min_rho_;

    for (int i=0; i < grid_size_*grid_size_ && max_rho_ == -1; i++)
    {
        if (host_rho_grid[i] > max_rho)
            max_rho = host_rho_grid[i];
    }

    if (max_rho_ == -1 && min_rho_ == -1)
    {
        if (max_rho - mean_rho > mean_rho - min_rho)
            min_rho = mean_rho - (max_rho - mean_rho);
        else
            max_rho = mean_rho - (min_rho - mean_rho);
    }

    if (max_rho_ != -1)
        max_rho = max_rho_;


    float* rho_grid_mapped_host = rho_grid_mapped_.HostData();
    for (int i=0; i < grid_size_; i++)
        for (int j=0; j < grid_size_; j++)
        {
            float rho;
            rho = host_rho_grid[grid_size_*j + i];
            if ((max_rho - min_rho) != 0)
                rho = (rho - min_rho) / (max_rho - min_rho);
            else
                rho = 0 * rho;

            // rho = (rho < 0) ? 0 : rho;
            // rho = (rho > 1) ? 1 : rho;

            rho_grid_mapped_host[grid_size_*3*j + 3*i + 0] = rho;
            rho_grid_mapped_host[grid_size_*3*j + 3*i + 1] = rho;
            rho_grid_mapped_host[grid_size_*3*j + 3*i + 2] = rho;
        }

    hipMemcpy(rho_grid_mapped_.Data(), rho_grid_mapped_host, 3*grid_size_*grid_size_*sizeof(float), hipMemcpyHostToDevice);
}

JFS_INLINE LBMSolverProps CudaLBMSolver::SolverProps()
{

    LBMSolverProps props{};
    props.grid_size = grid_size_;
    props.grid_length = grid_length_;
    props.btype = btype_;
    props.rho0 = rho0_;
    props.visc = visc_;
    props.lat_visc = lat_visc_;
    props.lat_tau = lat_tau_;
    props.uref = uref_;
    props.dt = dt_;
    props.rho_grid = rho_grid_.Data();
    props.f_grid = f_grid_.Data();
    props.f0_grid = f0_grid_.Data();
    props.u_grid = u_grid_.Data();
    props.force_grid = force_grid_.Data();
    props.failed_step = false;

    return props;
}

} // namespace jfs